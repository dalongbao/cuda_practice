#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main() {
	int c;
	int *dev_c;

	HANDLE_ERROR(hipMalloc( (void**)&dev_c, sizeof(int))); // allocates memory to the pointer dev_c

	add<<<1, 1>>>(1, 2, dev_c);
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost)); // copy contents of c to dev_c
	printf("%d\n", c);
	hipFree(dev_c);

	return 0;
}
