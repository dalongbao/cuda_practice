#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
#include "cpu_bitmap.h"

#define DIM 1000

// C
struct hipComplex {
	float r;
	float i;

	hipComplex(float a, float b) : r(a), i(b) {}

	__device__ float magnitude2() {return r * r + i * i}

	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}

	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r; i+a.i);
	}
};

int main() {
	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	dim3 grid(DIM, DIM);
	kernel<<<grid, 1>>>(dev_bitmap); 
	
	HANDLE_ERROR(hipMemcpy(
		bitmap.get_ptr(), 
		dev_bitmap, 
		bitmap.image_size(), 
		hipMemcpyDeviceToHost
	));

	bitmap.display_and_exit();
	HANDLE_ERROR(hipFree(dev_bitmap));
}

__global__ void kernel(unsigned char *ptr) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	int juliaValue = julia(x, y);
	ptr[offset*4] = 255 * juliaValue;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

__device__ void julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x)/(DIM / 2);
	float jy = scale * (float)(DIM / 2 - y)/(DIM / 2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i;
	for (i=0; i < 200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000) {
			return 0;
		}
	}
	return 1;
}

// __global__ void kernel(unsigned char *ptr) {
// 	for (int y=0; y < DIM; y++) {
// 		for (int x=0; x < DIM; x++) {
// 			int offset = x + y * DIM;
//
// 			int juliaValue = julia(x, y);
// 			ptr[offset*4 + 0] = 255 * juliaValue;
// 			ptr[offset*4 + 1] = 0;
// 			ptr[offset*4 + 2] = 0;
// 			ptr[offset*4 + 3] = 255;
// 		}
// 	}
// }

// int julia(int x, int y) {
// 	const float scale = 1.5;
// 	float jx = scale * (float)(DIM / 2 - x)/(DIM / 2);
// 	float jy = scale * (float)(DIM / 2 - y)/(DIM / 2);
//
// 	hipComplex c(-0.8, 0.156);
// 	hipComplex a(jx, jy);
//
// 	int i;
// 	for (i=0; i<200; i++) {
// 		a = a * a + c;
// 		if (a.magnitude2() > 1000) {
// 			return 0;
// 		}
// 	}
//
// 	return 1;
// }
