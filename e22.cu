#include "hip/hip_runtime.h"
#include <iostream>
#include "common/book.h"

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N*20)

// overlaps mean executing cuda kernels while perfoming a copy between device and host memory
// point of streams is to enable overlap of computation and data transfer
// order matters, don't block yourself. order them properly

__global__ void kernel(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;

		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;

		c[idx] = (as + bs) / 2;
	}
}

int main() {
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
	if (!prop.deviceOverlap) {
		printf("no overlaps means speedup from streams are gone :(");
		return 0;
	}

	hipEvent_t start, stop;
	float elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	hipStream_t stream0, stream1;
	HANDLE_ERROR(hipStreamCreate(&stream0)); // these always take the address because it allows cuda to directly operate on the memory location instead of taking a copy of the variable
	HANDLE_ERROR(hipStreamCreate(&stream1)); 

	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	HANDLE_ERROR(hipMalloc((void**)&dev_a0, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b0, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c0, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_a1, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b1, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c1, N*sizeof(int)));
	
	HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault)); // PINNED MEMORY!
	HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault)); // PINNING DOWN THE MEMORY??? freaky 
	HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

	for (int i=0; i < FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for (int i = 0; i < FULL_DATA_SIZE; i += N*2) { // copy memory to device
		HANDLE_ERROR(hipMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0));
		HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream1));
		HANDLE_ERROR(hipMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0));
		HANDLE_ERROR(hipMemcpyAsync(dev_b1, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream1));

		// kernel streams
		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		kernel<<<N/256, 256, 0, stream0>>>(dev_a1, dev_b1, dev_c1);

		// enqueue copies of c from device to pinned memory
		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1));
		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0));
	}
		
	HANDLE_ERROR(hipStreamSynchronize(stream0));
	HANDLE_ERROR(hipStreamSynchronize(stream1));
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("time taken %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipHostFree(host_a));
	HANDLE_ERROR(hipHostFree(host_b));
	HANDLE_ERROR(hipHostFree(host_c));

	HANDLE_ERROR(hipFree(dev_a0));
	HANDLE_ERROR(hipFree(dev_b0));
	HANDLE_ERROR(hipFree(dev_c0));
	HANDLE_ERROR(hipFree(dev_a1));
	HANDLE_ERROR(hipFree(dev_b1));
	HANDLE_ERROR(hipFree(dev_c1));
	HANDLE_ERROR(hipStreamDestroy(stream0));
	HANDLE_ERROR(hipStreamDestroy(stream1));
	return 0;
}
