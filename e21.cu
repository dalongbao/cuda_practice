#include "hip/hip_runtime.h"
#include <iostream>
#include "common/book.h"

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N*20)

// overlaps mean executing cuda kernels while perfoming a copy between device and host memory
// point of streams is to enable overlap of computation and data transfer

__global__ void kernel(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;

		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;

		c[idx] = (as + bs) / 2;
	}
}

int main() {
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
	if (!prop.deviceOverlap) {
		printf("no overlaps means speedup from streams are gone :(");
		return 0;
	}

	hipEvent_t start, stop;
	float elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	hipStream_t stream;
	HANDLE_ERROR(hipStreamCreate(&stream)); // these always take the address because it allows cuda to directly operate on the memory location instead of taking a copy of the variable

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));
	
	HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault)); // PINNED MEMORY!
	HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault)); // PINNING DOWN THE MEMORY??? freaky 
	HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

	for (int i=0; i < FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for (int i = 0; i < FULL_DATA_SIZE; i += N) { // copy memory to device
		HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream));
		HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream));
		kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N*sizeof(int), hipMemcpyDeviceToHost, stream));
	}
		
	HANDLE_ERROR(hipStreamSynchronize(stream));
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("time taken %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipHostFree(host_a));
	HANDLE_ERROR(hipHostFree(host_b));
	HANDLE_ERROR(hipHostFree(host_c));

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));
	HANDLE_ERROR(hipStreamDestroy(stream));
	return 0;
}
