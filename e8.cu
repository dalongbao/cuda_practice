#include "hip/hip_runtime.h"
#include <iostream>
#include "common/book.h"
#include "common/cpu_bitmap.h"

#define N 10

// inside angled brackets add<<<N, m>>>, N = no. blocks, m = thread per block
__global__ void add(int *a, int *b, int *c) {
	int tid = threadIdx.x;

	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

int main() {

	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	for (int i=0; i < N; i++) {
		a[i] = i;
		b[i] = i * i;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyDeviceToHost));

	add<<<1, N>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for (int i=0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i] );
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	// add<<<N, 1>>>(dev_a, dev_b, dev_c);

	return 0;
}
